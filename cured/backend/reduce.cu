#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "reduce.h"

namespace reduce
{

__global__ void ragged_array_row_wise_sum_reduce_kernel(const int *data, const int ra_size,
    const int *row_starts, const int *row_lengths, const int num_rows, int *out)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= num_rows)
  {
    return;
  }

  int row_start = row_starts[gid];
  int row_length = row_lengths[gid];

  for (int i = row_start; i < row_start+row_length; i++)
  {
    out[gid] += data[i];
  }
}

void ragged_array_row_wise_sum_reduce(const int *data, const int ra_size, 
    const int *row_starts, const int *row_lengths, const int num_rows, int *ret)
{
  int *data_d;
  int *row_starts_d;
  int *row_lengths_d;
  int *ret_d;

  cuda_errchk(hipMalloc(&data_d, sizeof(int)*ra_size));
  cuda_errchk(hipMalloc(&row_starts_d, sizeof(int)*num_rows));
  cuda_errchk(hipMalloc(&row_lengths_d, sizeof(int)*num_rows));
  cuda_errchk(hipMalloc(&ret_d, sizeof(int)*num_rows));

  cuda_errchk(hipMemcpy(data_d, data, sizeof(int)*ra_size, hipMemcpyHostToDevice));
  cuda_errchk(hipMemcpy(row_starts_d, row_starts, sizeof(int)*num_rows, hipMemcpyHostToDevice));
  cuda_errchk(hipMemcpy(row_lengths_d, row_lengths, sizeof(int)*num_rows, hipMemcpyHostToDevice));

  int thread_block_size = 128;
  int num_block = SDIV(num_rows, thread_block_size);
  ragged_array_row_wise_sum_reduce_kernel<<<num_block, thread_block_size>>>(
      data_d, ra_size, row_starts_d, row_lengths_d, num_rows, ret_d);
  hipDeviceSynchronize();

  cuda_errchk(hipMemcpy(ret, ret_d, sizeof(int)*num_rows, hipMemcpyDeviceToHost));

  cuda_errchk(hipFree(data_d));
  cuda_errchk(hipFree(row_starts_d));
  cuda_errchk(hipFree(row_lengths_d));
  cuda_errchk(hipFree(ret_d));
}

} // namespace reduce
